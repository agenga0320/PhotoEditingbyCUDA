#include "hip/hip_runtime.h"
#include <iostream>
#include "bmp_hdlr.h"
#include <hip/hip_runtime.h>
#include <>

#define NUM_THREADS 512

int canvas_r[bmp_size][bmp_size], canvas_g[bmp_size][bmp_size], canvas_b[bmp_size][bmp_size];
int h, w;
bool open = true;

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

__global__ static void monochrome(int* r,int* g, int* b ,int num ,int ht ,int wt)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / num;
	const int col = idx % num;
	if (row < ht && col < wt)
	{
		int y = (299 * r[row*num + col] + 587 * g[row*num + col] + 114 * b[row*num + col]) / 1000;
		r[row*num + col] = g[row*num + col] = b[row*num + col] = y;
	}
}

__global__ static void blur(int* r, int* g, int* b, int num, int ht ,int wt)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / num;
	const int col = idx % num;
	if (row < ht && col < wt)
	{
		int rr = 0, gg = 0, bb = 0, cnt = 0;
		for (int a = 0; a < 9; a++)
			for (int c = 0; c < 9; c++)
				if (row - 4 + a > -1 && row - 4 + a < ht && col - 4 + c > -1 && col - 4 + c < wt)
				{
					rr += r[(row - 4 + a) * num + col - 4 + c];
					gg += g[(row - 4 + a) * num + col - 4 + c];
					bb += b[(row - 4 + a) * num + col - 4 + c];
					cnt++;
				}
		r[row*num + col] = rr / cnt;
		g[row*num + col] = gg / cnt;
		b[row*num + col] = bb / cnt;
	}
}

__global__ static void moreblur(int* r, int* g, int* b, int num,int ht ,int wt)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / num;
	const int col = idx % num;
	if (row < ht && col < wt)
	{
		int rr = 0, gg = 0, bb = 0, cnt = 0, rrr = 0, ggg = 0, bbb = 0, x = 0;
		while (x < 4)
		{
			int i, j;
			if (x == 0) i = j = -5;
			else if (x == 1) i = 5;
			else if (x == 2) j = 5;
			else if (x == 3) i = -5;
			for (int a = 0; a < 9; a++)
				for (int c = 0; c < 9; c++)
					if (row - 4 + a+i > -1 && row - 4 + a+i < ht && col - 4 + c+j > -1 && col - 4 + c+j < wt)
					{
						rr += r[(row - 4 + a + i) * num + col - 4 + c + j];
						gg += g[(row - 4 + a + i) * num + col - 4 + c + j];
						bb += b[(row - 4 + a + i) * num + col - 4 + c + j];
						cnt++;
					}
			rrr += rr / cnt;
			ggg += gg / cnt;
			bbb += bb / cnt;
			x++;
		}
		r[row*num + col] = rrr / 4;
		g[row*num + col] = ggg / 4;
		b[row*num + col] = bbb / 4;
	}
}

__global__ static void focusblur(int* r, int* g, int* b, int num, int ht, int wt, int radius)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / num;
	const int col = idx % num;
	if (row < ht && col < wt)
	{
		const int dis = (row - radius)*(row - radius) + (col - radius)*(col - radius);
		const int diff = dis / (radius*radius);
		int rr = 0, gg = 0, bb = 0, cnt = 0;
		for (int ti = -diff; ti <= diff; ++ti) {
			for (int tj = -diff; tj <= diff; ++tj) {
				if (0 <= row + ti && row + ti < ht && 0 <= col + tj && col + tj < wt) {
					rr += r[(row + ti)*num + col + tj];
					gg += g[(row + ti)*num + col + tj];
					bb += b[(row + ti)*num + col + tj];
					++cnt;
				}
			}
		}
		r[row*bmp_size + col] = rr / cnt;
		g[row*bmp_size + col] = gg / cnt;
		b[row*bmp_size + col] = bb / cnt;
	}
}

__global__ static void smallsize(int* r, int* g, int* b, int num, int ht, int wt, float n)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / num;
	const int col = idx % num;
	if (row < ht/n && col < wt/n)
	{
		int pos = row * num * n + col * n;
		r[row*num + col] = r[pos];
		g[row*num + col] = g[pos];
		b[row*num + col] = b[pos];
	}
}

int main() {

	if (!open)
		return 0;

	if (!InitCUDA()) {
		return 0;
	}
	
	int *gpur, *gpug, *gpub;
	hipMalloc((void**)&gpur, sizeof(int) * bmp_size * bmp_size);
	hipMalloc((void**)&gpug, sizeof(int) * bmp_size * bmp_size);
	hipMalloc((void**)&gpub, sizeof(int) * bmp_size * bmp_size);
	
	hipMemcpy2D(gpur, sizeof(int) * bmp_size, canvas_r, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyHostToDevice);
	hipMemcpy2D(gpug, sizeof(int) * bmp_size, canvas_g, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyHostToDevice);
	hipMemcpy2D(gpub, sizeof(int) * bmp_size, canvas_b, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyHostToDevice);

	int i;
	std::cout << "what do tou want to do?\n1.monochrome\n2.blur\n3.more blur\n4.focus blur\n5.\n:";
	std::cin >> i;
	if (i == 1)
	{
		int blocks = (bmp_size + NUM_THREADS - 1) / NUM_THREADS;
		monochrome << <blocks * bmp_size, NUM_THREADS >> > (gpur, gpug, gpub, bmp_size,h ,w);
	}
	else if (i == 2)
	{
		int blocks = (bmp_size + NUM_THREADS - 1) / NUM_THREADS;
		blur << <blocks * bmp_size, NUM_THREADS >> > (gpur, gpug, gpub, bmp_size,h,w);
	}
	else if (i == 3)
	{
		int blocks = (bmp_size + NUM_THREADS - 1) / NUM_THREADS;
		moreblur << <blocks * bmp_size, NUM_THREADS >> > (gpur, gpug, gpub, bmp_size, h, w);
	}
	else if (i == 4)
	{
		int radius;
		if (h > w) radius = w / 3; else radius = h / 3;
		int blocks = (bmp_size + NUM_THREADS - 1) / NUM_THREADS;
		focusblur << <blocks * bmp_size, NUM_THREADS >> > (gpur, gpug, gpub, bmp_size, h, w, radius);
	}
	else if (i == 5)
	{
		float num = 1000;
		while (num > 100 || num < 1)
		{
		std::cout << "how many % smaller?(1~100):";
		std::cin >> num;
		}
		num = 100 / num;
		int blocks = (bmp_size + NUM_THREADS - 1) / NUM_THREADS;
		smallsize << <blocks * bmp_size, NUM_THREADS >> > (gpur, gpug, gpub, bmp_size, h, w, num);
		h /= num;
		w /= num;
		std::cout << h << " " << w << std::endl;
	}

	hipMemcpy2D(canvas_r, sizeof(int) * bmp_size, gpur, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyDeviceToHost);
	hipMemcpy2D(canvas_g, sizeof(int) * bmp_size, gpug, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyDeviceToHost);
	hipMemcpy2D(canvas_b, sizeof(int) * bmp_size, gpub, sizeof(int) * bmp_size, sizeof(int) * bmp_size, bmp_size, hipMemcpyDeviceToHost);


	return 0;
}
